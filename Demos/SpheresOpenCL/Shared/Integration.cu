#include "hip/hip_runtime.h"
/*
Bullet Continuous Collision Detection and Physics Library, http://bulletphysics.org
Copyright (C) 2006, 2007 Sony Computer Entertainment Inc. 

This software is provided 'as-is', without any express or implied warranty.
In no event will the authors be held liable for any damages arising from the use of this software.
Permission is granted to anyone to use this software for any purpose, 
including commercial applications, and to alter it and redistribute it freely, 
subject to the following restrictions:

1. The origin of this software must not be misrepresented; you must not claim that you wrote the original software. If you use this software in a product, an acknowledgment in the product documentation would be appreciated but is not required.
2. Altered source versions must be plainly marked as such, and must not be misrepresented as being the original software.
3. This notice may not be removed or altered from any source distribution.
*/

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <GL/glut.h>
#include <cuda_gl_interop.h>

#include "cutil_math.h"
#include "hip/hip_math_constants.h"

#include <hip/hip_vector_types.h>

//! Check for CUDA error
#define BT_GPU_CHECK_ERROR(errorMessage)									\
	do																		\
	{																		\
		hipError_t err = hipGetLastError();								\
		if(err != hipSuccess)												\
		{																	\
			fprintf(stderr,"Cuda error: %s in file '%s' in line %i : %s.\n",\
				errorMessage, __FILE__, __LINE__, hipGetErrorString( err));\
			btCuda_exit(EXIT_FAILURE);                                      \
		}                                                                   \
		err = hipDeviceSynchronize();                                      \
		if(err != hipSuccess)												\
		{																	\
			fprintf(stderr,"Cuda error: %s in file '%s' in line %i : %s.\n",\
				errorMessage, __FILE__, __LINE__, hipGetErrorString( err));\
			btCuda_exit(EXIT_FAILURE);										\
		}																	\
	}																		\
	while(0)


#define BT_GPU_SAFE_CALL_NO_SYNC(call)										\
	do																		\
	{																		\
		hipError_t err = call;												\
		if(err != hipSuccess)												\
		{																	\
			fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",	\
				__FILE__, __LINE__, hipGetErrorString( err) );             \
			btCuda_exit(EXIT_FAILURE);										\
		}																	\
	}																		\
	while(0)


#define BT_GPU_SAFE_CALL(call)												\
	do																		\
	{																		\
		BT_GPU_SAFE_CALL_NO_SYNC(call);										\
		hipError_t err = hipDeviceSynchronize();							\
		if(err != hipSuccess)												\
		{																	\
			fprintf(stderr,"Cuda errorSync in file '%s' in line %i : %s.\n",\
				__FILE__, __LINE__, hipGetErrorString( err) );				\
			btCuda_exit(EXIT_FAILURE);										\
		}																	\
	} while (0)


//Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}


__global__ void kIntegrateMotion(	float4* pPos,
									float4* pLinVel,
									int numObjects,
									float4* pParams, 
									float timeStep)
{
    uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numObjects) return;
	float4 pos = pPos[index];
	float4 linVel = pLinVel[index];
	float4 gravity = pParams[0];
	linVel += gravity * timeStep;
	pos += linVel * timeStep;
	pPos[index] = pos;
	pLinVel[index] = linVel;
}




extern "C"
{

void btCuda_exit(int val)
{
    fprintf(stderr, "Press ENTER key to terminate the program\n");
    getchar();
	exit(val);
}

void btCuda_allocateArray(void** devPtr, unsigned int size)
{
    BT_GPU_SAFE_CALL(hipMalloc(devPtr, size));
}

void btCuda_freeArray(void* devPtr)
{
    BT_GPU_SAFE_CALL(hipFree(devPtr));
}

void btCuda_copyArrayFromDevice(void* host, const void* device, unsigned int size)
{   
    BT_GPU_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}

void btCuda_copyArrayToDevice(void* device, const void* host, unsigned int size)
{
    BT_GPU_SAFE_CALL(hipMemcpy((char*)device, host, size, hipMemcpyHostToDevice));
}


void btCuda_registerGLBufferObject(unsigned int vbo)
{
    BT_GPU_SAFE_CALL(cudaGLRegisterBufferObject(vbo));
}

void* btCuda_mapGLBufferObject(unsigned int vbo)
{
    void *ptr;
    BT_GPU_SAFE_CALL(cudaGLMapBufferObject(&ptr, vbo));
    return ptr;
}

void btCuda_unmapGLBufferObject(unsigned int vbo)
{
    BT_GPU_SAFE_CALL(cudaGLUnmapBufferObject(vbo));
}


void btCuda_integrateMotion(void* pPos, 
							void* pLinVel, 
							int numObjects,
							void* pParams, 
							float timeStep)
{
    uint numThreads, numBlocks;
//    computeGridSize(numObjects, 256, numBlocks, numThreads);
    computeGridSize(numObjects, 128, numBlocks, numThreads);
    // execute the kernel
    kIntegrateMotion<<< numBlocks, numThreads >>>(	(float4*)pPos,
											(float4*)pLinVel,
											numObjects,
											(float4*)pParams,
											timeStep);
     BT_GPU_CHECK_ERROR("Kernel execution failed: kIntegrateMotion");
}


}